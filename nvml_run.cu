#include "hip/hip_runtime.h"
#include "nvml_run.h"
#include <stdio.h>
#include "results.h"
#include "debug.h"

nvmlReturn_t nvmlResult;
nvmlDevice_t nvmlDeviceID;
char deviceNameStr[128];

void nvml_setup(int device){

	// run the nvml Init phase
	nvmlResult = nvmlInit();
	if (NVML_SUCCESS != nvmlResult){
                printf("NVML init fail: %s\n", nvmlErrorString(nvmlResult));
                exit(0);
        }

	// get the Device ID string for NVML
	nvmlResult =  nvmlDeviceGetHandleByIndex(device, &nvmlDeviceID);
	if (NVML_SUCCESS != nvmlResult){
                printf("NVML get Device ID fail: %s\n", nvmlErrorString(nvmlResult));
                exit(0);
        }

	nvmlResult = nvmlDeviceGetName(nvmlDeviceID, deviceNameStr, sizeof(deviceNameStr)/sizeof(deviceNameStr[0]));
	if (NVML_SUCCESS != nvmlResult){
                printf("NVML get Device name fail: %s\n", nvmlErrorString(nvmlResult));
                exit(0);
        }


	//set the desired min and max GPU clock
	unsigned int gpu_clock;
	gpu_clock = assign_clock(deviceNameStr);
	nvmlResult = nvmlDeviceSetGpuLockedClocks(nvmlDeviceID, gpu_clock, gpu_clock);
	if (NVML_SUCCESS != nvmlResult){
                printf("NVML set GPU clock fail: %s\n", nvmlErrorString(nvmlResult));
                exit(0);
        }

	if(DEBUG){
		printf("----------- DEBUG -----------\n");
		printf("GPU name is %s;\n", deviceNameStr);
		printf("GPU clock set to %d;\n", gpu_clock);
		printf("-----------------------------\n");
	}

}

void nvml_reset(){

	nvmlResult = nvmlDeviceResetGpuLockedClocks(nvmlDeviceID);
	if (NVML_SUCCESS != nvmlResult){
                printf("NVML reset GPU fail: %s\n", nvmlErrorString(nvmlResult));
                exit(0);
        }

}

// function to set the near optimal frequency by device Name
unsigned int assign_clock(char *deviceName){
	unsigned int set_clock;
	if ( strcmp(deviceName, "Tesla V100-SXM2-32GB")  == 0 ){
		set_clock = 255;
	} 
	else {
		set_clock = 262;
	}

	return set_clock;
}

