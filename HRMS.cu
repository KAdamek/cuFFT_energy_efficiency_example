#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>


#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "debug.h"
#include "timer.h"
#include "utils_cuda.h"

#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>

#include "params.h"
#include "results.h"
#include "MSD_GPU_library.h"

#include "nvml_run.h"

#define PHS_NTHREADS 64
#define CT_CORNER_BLOCKS 1
#define CT_ROWS_PER_WARP 2
#define CT_NTHREADS 512
#define WARP 32

//----------------------------------------------------------------------------
//----------------------------------------------------------------------------

__global__ void GPU_simple_power_and_interbin_kernel(float2 *d_input_complex, float *d_output_power, int nTimesamples, float norm){
    int pos_x = blockIdx.x*blockDim.x + threadIdx.x;
    int pos_y = blockIdx.y*nTimesamples;
	
    float2 A;
    A.x = 0; A.y = 0;
	
    if( pos_x < nTimesamples ) {
		A = d_input_complex[pos_y + pos_x];
		d_output_power[pos_y + pos_x] = (A.x*A.x + A.y*A.y)*norm;
    }
}


__global__ void corner_turn_SM_kernel(float const* __restrict__ d_input, float *d_output, int primary_size, int secondary_size) {
    __shared__ float s_input[WARP*(WARP+1)*CT_CORNER_BLOCKS];
	
	int i, spos, itemp, pc, sc;
	size_t gpos;
	
	int warp_id = threadIdx.x>>5;
	int local_id = threadIdx.x & (WARP - 1);
	
	gpos=(size_t)((size_t)(blockIdx.y*(blockDim.x>>5)) + (size_t)warp_id)*CT_ROWS_PER_WARP*primary_size + (size_t)(blockIdx.x*CT_CORNER_BLOCKS*WARP) + (size_t)local_id;
	for(int by=0; by<CT_ROWS_PER_WARP; by++){
		spos=local_id*WARP + local_id + warp_id*CT_ROWS_PER_WARP + by;
		for(int bx=0; bx<CT_CORNER_BLOCKS; bx++){ // temporary 
			if(gpos<primary_size){
				s_input[spos]=d_input[gpos];
			}
			gpos=gpos + (size_t)WARP;
			spos=spos + WARP*(WARP+1);
		}
		gpos=gpos + (size_t)primary_size - (size_t)(CT_CORNER_BLOCKS*WARP);
	}
	
	__syncthreads();
	
	itemp=warp_id*CT_ROWS_PER_WARP*CT_CORNER_BLOCKS;
	for(i=0; i<CT_ROWS_PER_WARP*CT_CORNER_BLOCKS; i++){
		pc = (blockIdx.x*CT_CORNER_BLOCKS*WARP + itemp + i);
		sc = WARP*blockIdx.y + local_id;
		if( pc<primary_size && sc<secondary_size ) {
			gpos=(size_t)(pc*secondary_size) + (size_t)sc;
			spos=(itemp + i)*(WARP+1) + local_id;
			d_output[gpos]=s_input[spos];
		}
	}
}

__global__ void PHS_GPU_kernel(float const* __restrict__ d_input, float *d_output_SNR, ushort *d_output_harmonics, float *d_MSD, int nTimesamples, int nSpectra, int nHarmonics){
	float HS_value, temp_SNR, SNR;
	ushort max_SNR_harmonic;
	int pos;

	// reading 0th harmonic, i.e. fundamental frequency
	pos = blockIdx.x*nSpectra + blockIdx.y*blockDim.x + threadIdx.x;
	if( (blockIdx.y*blockDim.x + threadIdx.x)<nSpectra ){
		HS_value = __ldg(&d_input[pos]);
		SNR = (HS_value - __ldg(&d_MSD[0]))/(__ldg(&d_MSD[1]));
		max_SNR_harmonic = 0;
		
		if(blockIdx.x>0) {
			for(int f=1; f<nHarmonics; f++) {
				if( (blockIdx.x + f*blockIdx.x)<nTimesamples ) {
					pos = (blockIdx.x + f*blockIdx.x)*nSpectra + blockIdx.y*blockDim.x + threadIdx.x;
					HS_value = HS_value + __ldg(&d_input[pos]);
					temp_SNR = (HS_value - __ldg(&d_MSD[f*2]))/(__ldg(&d_MSD[2*f+1])); //assuming white noise 
					if(temp_SNR > SNR) {
						SNR = temp_SNR;
						max_SNR_harmonic = f;
					}
				}
			}
		}
		
		pos = blockIdx.x*nSpectra + blockIdx.y*blockDim.x + threadIdx.x;
		d_output_SNR[pos] = SNR;
		d_output_harmonics[pos] = max_SNR_harmonic;
	}
}
//----------------------------------------------------------------------------
//----------------------------------------------------------------------------



int Initiate_device(int device){
	int devCount;
	hipGetDeviceCount(&devCount);
	if(device<devCount) {
		hipSetDevice(device);
		return(0);
	}
	else return(1);	
}

int Check_free_memory(size_t total_input_FFT_size, size_t total_output_FFT_size){
	hipError_t err_code;
	size_t free_mem, total_mem;
	err_code = hipMemGetInfo(&free_mem,&total_mem);
	if(err_code!=hipSuccess) {
		printf("CUDA ERROR!\n");
		return(1);
	}
	
	if(free_mem<(total_input_FFT_size+total_output_FFT_size)) {
		printf("ERROR: Not enough GPU memory\n");
		return(1);
	}
	
	return(0);
}

double stdev(std::vector<double> *times, double mean_time){
	double sum = 0;
	for(size_t i=0; i<times->size(); i++){
		double x = (times->operator[](i)-mean_time);
		sum = sum + x*x;
	}
	double stdev = sqrt( sum/((double) times->size()) );
	return(stdev);
}

// ***********************************************************************************
// ***********************************************************************************
// ***********************************************************************************

int Calculate_GPU_HRMS(float2 *h_input, float *h_output, Performance_results *HRMS_results, int device){
	int nElements  = HRMS_results->nElements;
	int nHarmonics = HRMS_results->nHarmonics;
	int nSeries    = HRMS_results->nSeries;
	int nRuns      = HRMS_results->nRuns;
	size_t input_size = nElements*nSeries*sizeof(float2);
	size_t power_size = nElements*nSeries*sizeof(float);
	size_t output_size = nElements*nSeries;
	GpuTimer timer, total_timer;
	total_timer.Start();
	
	Initiate_device(device);
	
	float2 *d_input;
	float *d_power;
	float *d_output_SNR;
	ushort *d_output_harmonics;
	
	hipError_t err_code;
	err_code = hipMalloc((void **) &d_input, input_size);
	if(err_code!=hipSuccess) {
		printf("\nError in allocation of the device memory!\n");
		return(1);
	}
	err_code = hipMalloc((void **) &d_power, power_size);
	if(err_code!=hipSuccess) {
		printf("\nError in allocation of the device memory!\n");
		return(1);
	}
	err_code = hipMalloc((void **) &d_output_SNR, output_size*sizeof(float));
	if(err_code!=hipSuccess) {
		printf("\nError in allocation of the device memory!\n");
		return(1);
	}
	err_code = hipMalloc((void **) &d_output_harmonics, output_size*sizeof(ushort));
	if(err_code!=hipSuccess) {
		printf("\nError in allocation of the device memory!\n");
		return(1);
	}	
	
	err_code = hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
	if(err_code!=hipSuccess) {
		printf("\nError in allocation of the device memory!\n");
		return(1);
	}
	
	//-------------------- cuFFT ----------------->
	hipfftHandle plan;
	hipfftResult cuFFT_error;
	cuFFT_error = hipfftPlan1d(&plan, nElements, HIPFFT_C2C, nSeries);
	double FFT_execution_time = 0;
	if (HIPFFT_SUCCESS == cuFFT_error) {
		nvml_setup(device);
		for(int f=0; f<nRuns; f++){
			timer.Start();
			hipfftExecC2C(plan, (hipfftComplex *) d_input, (hipfftComplex *) d_input, HIPFFT_FORWARD);
			timer.Stop();
			FFT_execution_time += timer.Elapsed();
		}
		// stop before reset to default; kernel call is async
		hipDeviceSynchronize();
		nvml_reset();
		FFT_execution_time = FFT_execution_time/((double) nRuns);
		HRMS_results->GPU_FFT_time = FFT_execution_time;
	}
	else printf("CUFFT error: Plan creation failed");
	hipfftDestroy(plan);
	//--------------------------------------------<
	
	//------------- Power calculation ------------>
    int power_blocks_x, power_blocks_y;
	
    power_blocks_x = (nElements + 256 - 1)/256;
    power_blocks_y = nSeries;
	
    dim3 power_blockDim(256, 1, 1);
    dim3 power_gridSize(power_blocks_x ,power_blocks_y , 1);
	
	FFT_execution_time = 0;
	for(int f=0; f<nRuns; f++){
		timer.Start();
		GPU_simple_power_and_interbin_kernel<<< power_gridSize , power_blockDim >>>(d_input, d_power, nElements, 1);	
		timer.Stop();
		FFT_execution_time += timer.Elapsed();
	}
	FFT_execution_time = FFT_execution_time/((double) nRuns);
	HRMS_results->GPU_MSD_time = FFT_execution_time;	
	//--------------------------------------------<	

	//--------------------- MSD ------------------>
	int nBatches = 1;
	int MSD_size = MSD_RESULTS_SIZE*nBatches*sizeof(float);
	int MSD_elements_size = nBatches*sizeof(size_t);
	
	float *d_MSD;
	size_t *d_MSD_nElements;
	if ( hipSuccess != hipMalloc((void **) &d_MSD, MSD_size)) {
		printf("CUDA API error while allocating GPU memory\n");
	}
	if ( hipSuccess != hipMalloc((void **) &d_MSD_nElements, MSD_elements_size)) {
		printf("CUDA API error while allocating GPU memory\n");
	}
	
	MSD_Error MSD_error;
	MSD_Configuration MSD_conf;
	std::vector<size_t> dimensions={ (size_t) nSeries, (size_t) nElements};
	bool outlier_rejection = false;
	int offset = 0;
	double outlier_rejection_sigma = 3.0;
	MSD_error = MSD_conf.Create_MSD_Plan(dimensions, offset, outlier_rejection, outlier_rejection_sigma, nBatches);
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	FFT_execution_time = 0;
	for(int f=0; f<nRuns; f++){
		timer.Start();
		MSD_error = MSD_GetMeanStdev(d_MSD, d_MSD_nElements, d_power, MSD_conf);
		timer.Stop();
		FFT_execution_time += timer.Elapsed();
	}
	FFT_execution_time = FFT_execution_time/((double) nRuns);
	HRMS_results->GPU_MSD_time += FFT_execution_time;
	if(MSD_error!=MSDSuccess) Get_MSD_Error(MSD_error);
	//--------------------------------------------<	
	
	//--------------- Harmonic Sum --------------->
	int CT_nBlocks_x, CT_nBlocks_y;
	int Elements_per_block=CT_CORNER_BLOCKS*WARP;
	CT_nBlocks_x = (nElements + Elements_per_block - 1)/Elements_per_block;
	CT_nBlocks_y = (nSeries + WARP + 1)/WARP;
	dim3 CT_gridSize(CT_nBlocks_x, CT_nBlocks_y, 1);
    dim3 CT_blockSize(CT_NTHREADS, 1, 1);
	FFT_execution_time = 0;
	for(int f=0; f<nRuns; f++){
		timer.Start();
		corner_turn_SM_kernel<<< CT_gridSize, CT_blockSize >>>(d_power, (float *) d_input, nElements, nSeries);
		timer.Stop();
		FFT_execution_time += timer.Elapsed();
	}
	FFT_execution_time = FFT_execution_time/((double) nRuns);
	HRMS_results->GPU_HRMS_time = FFT_execution_time;	
	
	
	int HRMS_nBlocks_x, HRMS_nBlocks_y;
	HRMS_nBlocks_x = nElements;
    HRMS_nBlocks_y = (nSeries + PHS_NTHREADS - 1)/PHS_NTHREADS;
    dim3 HRMS_gridSize(HRMS_nBlocks_x, HRMS_nBlocks_y, 1);
    dim3 HRMS_blockSize(PHS_NTHREADS, 1, 1);
	
	FFT_execution_time = 0;
	for(int f=0; f<nRuns; f++){
		timer.Start();
		PHS_GPU_kernel<<< HRMS_gridSize, HRMS_blockSize >>>((float *) d_input, d_output_SNR, d_output_harmonics, d_MSD, nElements, nSeries, nHarmonics);
		timer.Stop();
		FFT_execution_time += timer.Elapsed();
	}
	FFT_execution_time = FFT_execution_time/((double) nRuns);
	HRMS_results->GPU_HRMS_time += FFT_execution_time;
	//--------------------------------------------<
	
	total_timer.Stop();
	HRMS_results->GPU_total_time = HRMS_results->GPU_HRMS_time + HRMS_results->GPU_MSD_time + HRMS_results->GPU_FFT_time;

	hipFree(d_MSD);
	hipFree(d_MSD_nElements);
	hipFree(d_input);
	hipFree(d_power);
	hipFree(d_output_SNR);
	hipFree(d_output_harmonics);
	
	return(0);
}
